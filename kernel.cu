#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <math.h>

#include <hip/hip_runtime_api.h>
#ifdef __HIPCC__
#define cuda_SYNCTHREADS() __syncthreads()
#else
#define cuda_SYNCTHREADS()
#endif
#define BLOQUE 1
#define HILOS 512

__device__ float calcularArea(float inicio, float final, float base)
{
    float medio = (inicio + final) / 2;
    float altura = 4 / (1 + (medio) * (medio));
    return base * altura;
}

__global__ void calcularPi(float* pi, int* precisaoEscolhida)
{
    int lancamentos = *precisaoEscolhida;
    int identificador = threadIdx.x;
    float inicio;
    float final;
    extern __shared__ float area[];
    float superVar = 1 / (float)lancamentos;
    cuda_SYNCTHREADS();

    int* array;
    if (lancamentos % 2 != 0)
    {
        if (identificador == (lancamentos - 1))
        {
            array[0] += array[identificador];
        }
    }
    cuda_SYNCTHREADS();

    int salto = lancamentos / 2;

    while (salto)
    {
        if (identificador < salto)
        {
            area[identificador] = array[0] + array[identificador];
        }
    }
    cuda_SYNCTHREADS();

    if (identificador == 0)
    {
        *pi = area[identificador];
    }
}

__host__ int clean_stdin(void)
{
    while (getchar() != '\n')
        ;
    return 1;
}

int main(int argc, char** argv)
{
    float* dev_pi, * hst_pi;
    int* dev_precition;
    hipSetDevice(0);        
    hipEvent_t start, stop; 
    int precition;
    char c;
    char linea[] = "---------------------------------------------------------------------";
    hipDeviceProp_t features;               
    hipGetDeviceProperties(&features, 0);
    
    do
    {
        printf("Qual precisao deseja para calcular PI, maximo %d: ", features.maxThreadsPerBlock);
        //
        if (scanf("%d%c", &precition, &c) != 2 || c != '\n')
        {
            printf("Valor invalido\n");
            clean_stdin();
        }
    } while (precition < 0 || precition > features.maxThreadsPerBlock);
    //Reservando memoria
    hst_pi = (float*)malloc(sizeof(float));
    hipMalloc((void**)&dev_pi, sizeof(float));
    hipMalloc((void**)&dev_precition, sizeof(int));
    hipMemcpy(dev_precition, &precition, sizeof(int), hipMemcpyHostToDevice);
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    calcularPi <<< BLOQUE, precition, precition * sizeof(float) >>> (dev_pi, dev_precition);
    hipMemcpy(hst_pi, dev_pi, sizeof(float), hipMemcpyDeviceToHost);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    //Impressao dos resultados
    printf("\n%s\nmemoria compartilhada disponivel %d KiB:\n", linea, features.sharedMemPerBlock / 1024);
    printf("Valo de PI calculado: %f\n", *hst_pi);
    printf("Tempo de execucao \n", elapsedTime);
    getchar();
    return 0;
}
